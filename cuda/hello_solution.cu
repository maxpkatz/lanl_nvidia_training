
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello() {
    printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

int main() {
    hello<<<2, 2>>>();
    hipDeviceSynchronize();
}
