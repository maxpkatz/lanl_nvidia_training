
#include <hip/hip_runtime.h>
#include <stdio.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

const int DSIZE = 4096;
const int block_size = 256;  // CUDA maximum is 1024

// Vector add kernel: B = B + A
__global__ void vadd(const float *A, float *B, int ds) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // create typical 1D thread index from built-in variables
    if (idx < ds) {
        B[idx] = B[idx] + A[idx];                    // do the vector (element) add here
    }
}

int main() {
    float *A, *B;
    hipMallocManaged(&A, DSIZE * sizeof(float)); // allocate device space for vector A
    hipMallocManaged(&B, DSIZE * sizeof(float)); // allocate device space for vector B
    cudaCheckErrors("Memory allocation failure"); // error checking

    // Initialize vectors
    for (int i = 0; i < DSIZE; ++i) {
        A[i] = rand() / (float) RAND_MAX;
        B[i] = 0.0;
    }

    // Launch kernel to do the vector addition
    vadd<<<(DSIZE + block_size - 1) / block_size, block_size>>>(A, B, DSIZE);
    cudaCheckErrors("Kernel launch failure");

    // Wait for kernel to complete; the return code from
    // cudaDeviceSynchronize() indicates whether there were any
    // runtime errors during kernel execution.
    hipDeviceSynchronize();
    cudaCheckErrors("Kernel execution failure");

    // Verify on host that all values in A are the same as B
    for (int i = 0; i < DSIZE; ++i) {
        if (A[i] != B[i]) {
            printf("Error, A[%d] != B[%d]\n", i, i);
            return -1;
        }
    }

    printf ("Success!\n");

    return 0;
}
